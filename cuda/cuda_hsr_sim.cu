#include "hip/hip_runtime.h"
// Parses Reliquary Archiver JSON file and fills in information for HSR relics and characters
// Reliquary repository found at: https://github.com/IceDynamix/reliquary-archiver/tree/main
// The parser itself is simple and mainly focuses on converting JSON file to structs to work with in C++.

// Process in 3 steps:
// 1. Parellel Scan JSON file to generate structural index arrays
// 2. Parellel parse JSON file based on structural index arrays to generate HSR Relic data
// 3. Parellel generate HSR Relic data to HSR Sim data

// includes and defines
#include <iostream>
#include <fstream>
#include <string>
#include <map>
#include "cuda_hsr_sim.h"

#define FILEPATH "/home/hapuum/cuda_learn/resource/relic_data.json"
#define TEST_FILEPATH "/home/hapuum/cuda_learn/resource/test.json"
#define CHAR_PER_THREAD 4096        // 1 full memory page
#define NUM_PER_BLOCK 256           // makes 1 block = 1MiB
#define T0_DEFAULT_MAX_SIZE 32768  // default size of DeviceVector at thread 0 vs rest.
#define REST_DEFAULT_MAX_SIZE 64  // thread 0 is reserved more space for reduction stage

typedef struct Payload
{
    JsonObject json;
    std::string type;
};

class
JsonObject
{
    std::map<std::string, Payload> map;
    
    public:
        JsonObject() {}
        ~JsonObject() {}

    JsonObject operator[](std::string s) {
        return map[s].json;
    }

    void addEntry(int data) {};
    void addEntry(bool data) {};
    void addEntry(std::string data) {};
    void addEntry(JsonObject data[]) {};
    void addEntry(JsonObject data) {};

};

template<typename T>
class
DeviceVector 
{
    T* data;
    size_t capacity, length;
    public:
        __device__ DeviceVector() : data(nullptr), capacity(16), length(0) 
        {
            data = new T[capacity];
        }
        __device__ DeviceVector(size_t c) : data(nullptr), capacity(c), length(0) 
        {
            data = new T[capacity];
        }

        __device__ ~DeviceVector() { delete[] data; }

        __device__ void push_back(const T& value) 
        {
            if (length >= capacity) 
            {
                // Expand capacity
                capacity *= 2;
                T* new_data = new T[capacity];
                for (size_t i = 0; i < length; ++i) new_data[i] = data[i];
                delete[] data;
                data = new_data;
            }
            data[length++] = value;
        }

        __device__ void set(size_t idx, const T& value) { data[idx] = value; }

        __device__ void join(DeviceVector<T>* next) 
        {
            for (int i = 0; i < next->length; i++) {
                this->push_back(next->get(i));
            }
        }

        __device__ T* getInternalArray() {
            return data;
        }

        __host__ __device__ void printVector() {
            for (int i = 0; i < this->length; i++) {
                printf("%d \n", this->get(i));
            }
            printf("\n");
        }

        // host can only read and cannot modify.
        __host__ __device__ T& operator[](size_t idx) { return data[idx]; }
        __host__ __device__ size_t size() const { return length; }
        __host__ __device__ T& get(size_t idx) { return data[idx]; }

};

__global__
void
build_structural_index
(
    // input
    char* file_data, 
    size_t file_size,
    // output
    int* open_brace_positions,
    int* close_brace_positions,
    int* open_bracket_positions,
    int* close_bracket_positions,
    int* colon_positions,
    int* comma_positions,
    size_t* size_open_brace,
    size_t* size_close_brace,
    size_t* size_open_bracket,
    size_t* size_close_bracket,
    size_t* size_colon,
    size_t* size_comma
) 
{

    size_t vector_capacity = (threadIdx.x == 0) ? T0_DEFAULT_MAX_SIZE : REST_DEFAULT_MAX_SIZE;

    // @TODO: MAYBE Refactor DeviceVector to __shared__ priorityqueue interface, 
    //          where each thread adds to the shared priorityqueue in some safe locked manner
    //          Such implementation can remove 1) reduction stage, 2) sorting stage for pair of symbols
    //          This might require coming up with a struct/class that can be added to this priorityqueue, 
    //          that can be either: pair {} or pair [] or : or ,
    
    DeviceVector<int>* open_brace_vector = new DeviceVector<int>(vector_capacity);
    DeviceVector<int>* close_brace_vector = new DeviceVector<int>(vector_capacity);
    DeviceVector<int>* open_bracket_vector = new DeviceVector<int>(vector_capacity);
    DeviceVector<int>* close_bracket_vector = new DeviceVector<int>(vector_capacity);
    DeviceVector<int>* colon_vector = new DeviceVector<int>(vector_capacity);
    DeviceVector<int>* comma_vector = new DeviceVector<int>(vector_capacity);

    // iterate through each character of assigned section of file_data
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    for (int i = 0; i < CHAR_PER_THREAD; i++) 
    {
        char c = file_data[idx * CHAR_PER_THREAD + i];
        switch (c) 
        {
            case '{':
                open_brace_vector->push_back(idx * CHAR_PER_THREAD + i);
                break;
            case '}':
                close_brace_vector->push_back(idx * CHAR_PER_THREAD + i);
                break;
            case '[':
                open_bracket_vector->push_back(idx * CHAR_PER_THREAD + i);
                break;
            case ']':
                close_bracket_vector->push_back(idx * CHAR_PER_THREAD + i);
                break;
            case ':':
                colon_vector->push_back(idx * CHAR_PER_THREAD + i);
                break;
            case ',':
                comma_vector->push_back(idx * CHAR_PER_THREAD + i);
                break;
        }
    }
    
    __syncthreads();
    
    // join vectors by reduction
    // initialize 2D array of pointers to vectors storing each thread's vectors

    __shared__ DeviceVector<int>* vector_array[NUM_PER_BLOCK][6];
    vector_array[threadIdx.x][0] = open_brace_vector;
    vector_array[threadIdx.x][1] = close_brace_vector;
    vector_array[threadIdx.x][2] = open_bracket_vector;
    vector_array[threadIdx.x][3] = close_bracket_vector;
    vector_array[threadIdx.x][4] = colon_vector;
    vector_array[threadIdx.x][5] = comma_vector;

    for (int step = 1; step < blockDim.x; step *= 2) 
    {
        if (threadIdx.x % (2 * step) == 0) 
        {
            for (int i = 0; i < 6; i++) 
            {
                vector_array[threadIdx.x][i]->join(vector_array[threadIdx.x + step][i]);
            }
        }
        __syncthreads();
    }
    
    // assign return values
    // work 6 threads for copying each array to output 
    switch (threadIdx.x) 
    {
        case 0:
            for (int i = 0; i < vector_array[0][threadIdx.x]->size(); i++) 
            {
                open_brace_positions[i] = vector_array[0][threadIdx.x]->get(i);
            }
            *size_open_brace = vector_array[0][threadIdx.x]->size();
            break;
        case 1:
            for (int i = 0; i < vector_array[0][threadIdx.x]->size(); i++) 
            {
                close_brace_positions[i] = vector_array[0][threadIdx.x]->get(i);
            }
            *size_close_brace = vector_array[0][threadIdx.x]->size();
            break;
        case 2:
            for (int i = 0; i < vector_array[0][threadIdx.x]->size(); i++) 
            {
                open_bracket_positions[i] = vector_array[0][threadIdx.x]->get(i);
            }
            *size_open_bracket = vector_array[0][threadIdx.x]->size();
            break;
        case 3:
            for (int i = 0; i < vector_array[0][threadIdx.x]->size(); i++) 
            {
                close_bracket_positions[i] = vector_array[0][threadIdx.x]->get(i);
            }
            *size_close_bracket = vector_array[0][threadIdx.x]->size();
            break;
        case 4:
            for (int i = 0; i < vector_array[0][threadIdx.x]->size(); i++) 
            {
                colon_positions[i] = vector_array[0][threadIdx.x]->get(i);
            }
            *size_colon = vector_array[0][threadIdx.x]->size();
            break;
        case 5:
            for (int i = 0; i < vector_array[0][threadIdx.x]->size(); i++) 
            {
                comma_positions[i] = vector_array[0][threadIdx.x]->get(i);
            }
            *size_comma = vector_array[0][threadIdx.x]->size();
            break;
        default: 
            break;
    }
}

int main() {
    std::cout << "HSR Reliquary Archiver JSON Parser" << std::endl;
    // load json file
    bool testing = 0;
    std::string filepath = (testing) ? TEST_FILEPATH : FILEPATH;

    std::ifstream file(filepath);
    if (!file.is_open()) 
    {
        std::cerr << "Error opening file: " << filepath << std::endl;
        return EXIT_FAILURE;
    }
    std::string json_content((std::istreambuf_iterator<char>(file)), std::istreambuf_iterator<char>());
    file.close();
    std::cout << "File loaded successfully. Size: " << json_content.size() << " bytes" << std::endl;

    // scan json file to generate structural indices
    char* d_json_content;
    
    int* openBracePositions      = (int*)malloc(sizeof(int) * T0_DEFAULT_MAX_SIZE);
    int* closeBracePositions     = (int*)malloc(sizeof(int) * T0_DEFAULT_MAX_SIZE);
    int* openBracketPositions    = (int*)malloc(sizeof(int) * T0_DEFAULT_MAX_SIZE);
    int* closeBracketPositions   = (int*)malloc(sizeof(int) * T0_DEFAULT_MAX_SIZE);
    int* colonPositions          = (int*)malloc(sizeof(int) * T0_DEFAULT_MAX_SIZE);
    int* commaPositions          = (int*)malloc(sizeof(int) * T0_DEFAULT_MAX_SIZE);

    size_t size_openBrace_vector;
    size_t size_closeBrace_vector;
    size_t size_openBracket_vector;
    size_t size_closeBracket_vector;
    size_t size_colon_vector;
    size_t size_comma_vector;

    int* d_openBracePositions;
    int* d_closeBracePositions;
    int* d_openBracketPositions;
    int* d_closeBracketPositions;
    int* d_colonPositions;
    int* d_commaPositions;

    size_t* d_size_openBrace_vector;
    size_t* d_size_closeBrace_vector;
    size_t* d_size_openBracket_vector;
    size_t* d_size_closeBracket_vector;
    size_t* d_size_colon_vector;
    size_t* d_size_comma_vector;

    size_t json_size = json_content.size();

    int num_block = (json_size + NUM_PER_BLOCK * CHAR_PER_THREAD) / ((NUM_PER_BLOCK) * (CHAR_PER_THREAD));

    hipMalloc((void**) &d_json_content, json_size);
    hipMalloc((void**) &d_openBracePositions, sizeof(int) * T0_DEFAULT_MAX_SIZE);
    hipMalloc((void**) &d_openBracketPositions, sizeof(int) * T0_DEFAULT_MAX_SIZE);
    hipMalloc((void**) &d_closeBracePositions, sizeof(int) * T0_DEFAULT_MAX_SIZE);
    hipMalloc((void**) &d_closeBracketPositions, sizeof(int) * T0_DEFAULT_MAX_SIZE);
    hipMalloc((void**) &d_colonPositions, sizeof(int) * T0_DEFAULT_MAX_SIZE);
    hipMalloc((void**) &d_commaPositions, sizeof(int) * T0_DEFAULT_MAX_SIZE);

    hipMalloc((void**)&d_size_openBrace_vector, sizeof(size_t));
    hipMalloc((void**)&d_size_closeBrace_vector, sizeof(size_t));
    hipMalloc((void**)&d_size_openBracket_vector, sizeof(size_t));
    hipMalloc((void**)&d_size_closeBracket_vector, sizeof(size_t));
    hipMalloc((void**)&d_size_colon_vector, sizeof(size_t));
    hipMalloc((void**)&d_size_comma_vector, sizeof(size_t));

    hipMemcpy(d_json_content, json_content.data(), json_size, hipMemcpyHostToDevice);

    build_structural_index<<<num_block, NUM_PER_BLOCK>>> (
        d_json_content,
        json_size,    
        d_openBracePositions,  
        d_closeBracePositions,
        d_openBracketPositions,
        d_closeBracketPositions,
        d_colonPositions,
        d_commaPositions,
        d_size_openBrace_vector,
        d_size_closeBrace_vector,
        d_size_openBracket_vector,
        d_size_closeBracket_vector,
        d_size_colon_vector,
        d_size_comma_vector
    );
    
    hipMemcpy(openBracePositions, d_openBracePositions, sizeof(int) * T0_DEFAULT_MAX_SIZE, hipMemcpyDeviceToHost);
    hipMemcpy(closeBracePositions, d_closeBracePositions, sizeof(int) * T0_DEFAULT_MAX_SIZE, hipMemcpyDeviceToHost);
    hipMemcpy(openBracketPositions, d_openBracketPositions, sizeof(int) * T0_DEFAULT_MAX_SIZE, hipMemcpyDeviceToHost);
    hipMemcpy(closeBracketPositions, d_closeBracketPositions, sizeof(int) * T0_DEFAULT_MAX_SIZE, hipMemcpyDeviceToHost);
    hipMemcpy(colonPositions, d_colonPositions, sizeof(int) * T0_DEFAULT_MAX_SIZE, hipMemcpyDeviceToHost);
    hipMemcpy(commaPositions, d_commaPositions, sizeof(int) * T0_DEFAULT_MAX_SIZE, hipMemcpyDeviceToHost);

    hipMemcpy(&size_openBrace_vector, d_size_openBrace_vector, sizeof(size_t), hipMemcpyDeviceToHost);
    hipMemcpy(&size_closeBrace_vector, d_size_closeBrace_vector, sizeof(size_t), hipMemcpyDeviceToHost);
    hipMemcpy(&size_openBracket_vector, d_size_openBracket_vector, sizeof(size_t), hipMemcpyDeviceToHost);
    hipMemcpy(&size_closeBracket_vector, d_size_closeBracket_vector, sizeof(size_t), hipMemcpyDeviceToHost);
    hipMemcpy(&size_colon_vector, d_size_colon_vector, sizeof(size_t), hipMemcpyDeviceToHost);
    hipMemcpy(&size_comma_vector, d_size_comma_vector, sizeof(size_t), hipMemcpyDeviceToHost);

    for (int i = 0; i < size_openBrace_vector; i++) {
        std::cout << openBracePositions[i] << std::endl;
    }


    std::cout << "size of open brace vector: " << size_openBrace_vector << std::endl;

    if (size_openBrace_vector != size_closeBrace_vector) {
        std::cout << "invalid file content: { and } amount does not match" << std::endl;
        return EXIT_FAILURE;
    }

    if (size_openBracket_vector != size_closeBracket_vector) {
        std::cout << "invalid file content: [ and ] amount does not match" << std::endl;
        return EXIT_FAILURE;
    }

    size_t pos_CHARACTERS   = json_content.find("characters");
    size_t pos_RELICS       = json_content.find("relics");

    if (pos_CHARACTERS == std::string::npos || pos_RELICS == std::string::npos) {
        std::cout << "invalid file content: cannot find \"characters\" or \"relics\"" << std::endl;
        return EXIT_FAILURE;
    }

    std::cout << "\"characters\" is found at: " << pos_CHARACTERS << std::endl;
    std::cout << "\"relics\" is found at: " << pos_RELICS << std::endl;

    // DO SOME PROCESSING TO FIND WHERE THE RANGE FOR THESE END?

    //     struct Task {
    //     int start;
    //     int end;
    //      };

    // __device__ Task workQueue[MAX_TASKS];
    // __device__ int queueHead = 0;

    // __global__ void parse_json(char* json, int* openBraces, int* closeBraces, int numObjects) {
    //     int tid = threadIdx.x + blockIdx.x * blockDim.x;
    //     while (true) {
    //         int taskIdx = atomicAdd(&queueHead, 1);
    //         if (taskIdx >= numObjects) break;
    //         Task t = workQueue[taskIdx];
    //         // Parse json[t.start ... t.end]
               // SEARCH FOR "VARIABLE"
               // SWITCH("VARIABLE"):
               //   case("characters"), case("relics"), remaining things we can simply skip.
    //         // If nested object found, atomicAdd to queueHead and add new Task
    //     }
    // }



    return EXIT_SUCCESS;
}