
#include <hip/hip_runtime.h>
#include <stdio.h>

// Calculates the Matric multiplication of first * second, stores into result
// first matrix has dimensions N x M
// second matrix has dimensions M x L
// result matrix has dimensions N x L
__global__ void MatMul(int first[], int second[], int result[], int N, int M, int L) {
    int row = threadIdx.x;
    int col = threadIdx.y;
    
    int sum = 0;
    for (int k = 0; k < M; k++) {
        sum += first[row * M + k] * second[k * L + col];
    }

    result[row * L + col] = sum;
    printf("Row: %d, Col: %d, Value: %d\n", row, col, sum);
}

int main() {
    // USER SHOULD ONLY CHANGE THIS PART
    int A_rows = 3, A_cols = 2;
    int B_rows = 2, B_cols = 2;
    int* A = new int[A_rows * A_cols] {1, 2, 3, 4, 5, 6};
    int* B = new int[B_rows * B_cols] {1, 2, 3, 4};
    // END OF USER CHANGEABLE PART

    if (A_cols != B_rows) {
        printf("Incompatible matrix dimensions for multiplication.\n");
        return EXIT_FAILURE;
    }

    int *d_A, *d_B, *result;
    hipMalloc((void**)&d_A,    A_rows * A_cols * sizeof(int));
    hipMalloc((void**)&d_B,    B_rows * B_cols * sizeof(int));
    hipMalloc((void**)&result, A_rows * B_cols * sizeof(int));
    

    hipMemcpy(d_A, A, 3 * 2 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, 2 * 2 * sizeof(int), hipMemcpyHostToDevice);

    dim3 numBlocks          = dim3(1, 1, 1);
    dim3 threadsPerBlock    = dim3(A_rows, B_cols, 1);

    MatMul<<<numBlocks, threadsPerBlock>>>(d_A, d_B, result, A_rows, A_cols, B_cols);
    
    hipDeviceSynchronize();

    int* resultHost = new int[A_rows * B_cols];
    hipMemcpy(resultHost, result, A_rows * B_cols * sizeof(int), hipMemcpyDeviceToHost);
    printf("Result matrix:\n");
    for (int i = 0; i < A_rows; i++) {
        for (int j = 0; j < B_cols; j++) {
            printf("%d ", resultHost[i * 2 + j]);
        }
        printf("\n");
    }

    return EXIT_SUCCESS;
}